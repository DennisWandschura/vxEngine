#undef __INTEL_COMPILER
#define _VX_CUDA

#include <thrust/sort.h>
#include <Windows.h>
#include <cuda_gl_interop.h>
#include <thrust/device_ptr.h>

struct __builtin_align__(16) CompressedRay
{
	uint32_t mortonCode;
	uint32_t compressedRay[3];

	__device__ friend bool operator<(const CompressedRay &l, const CompressedRay &r)
	{
		return l.mortonCode < r.mortonCode;
	}
};

struct __builtin_align__(8) RayLink
{
	uint32_t mortonCode;
	uint32_t rayIndex;

	__device__ friend bool operator<(const RayLink &l, const RayLink &r)
	{
		return l.mortonCode < r.mortonCode;
	}
};

hipGraphicsResource_t g_rayLinkBuffer{nullptr};
hipGraphicsResource_t g_rayListBuffer{ nullptr };

void registerRayLinkBuffer(uint32_t rayLinkBufferId)
{
	hipGraphicsGLRegisterBuffer(&g_rayLinkBuffer, rayLinkBufferId, hipGraphicsRegisterFlagsNone);
}

void registerRayListBuffer(uint32_t rayListBufferId)
{
	hipGraphicsGLRegisterBuffer(&g_rayListBuffer, rayListBufferId, hipGraphicsRegisterFlagsNone);
}

void initializeCUDA(uint32_t rayLinkBufferId, uint32_t rayListBufferId)
{
	registerRayLinkBuffer(rayLinkBufferId);
	registerRayListBuffer(rayListBufferId);
}

void shutdownCUDA()
{
	hipGraphicsUnregisterResource(g_rayLinkBuffer);
	hipGraphicsUnregisterResource(g_rayListBuffer);

	g_rayLinkBuffer = nullptr;
	g_rayListBuffer = nullptr;
}

void cudaSortRayLinks(uint32_t rayLinkCount)
{
	hipGraphicsMapResources(1, &g_rayListBuffer);

	size_t sizeInBytes = 0;
	void* rayLinkPtr = nullptr;
	hipGraphicsResourceGetMappedPointer(&rayLinkPtr, &sizeInBytes, g_rayLinkBuffer);

	thrust::device_ptr<RayLink> devicePtr((RayLink*)rayLinkPtr);

	thrust::sort(devicePtr, devicePtr + rayLinkCount);

	hipGraphicsUnmapResources(1, &g_rayLinkBuffer);
}

void cudaSortRayList(uint32_t rayCount)
{
	hipGraphicsMapResources(1, &g_rayListBuffer);

	size_t sizeInBytes = 0;
	void* rayListPtr = nullptr;
	hipGraphicsResourceGetMappedPointer(&rayListPtr, &sizeInBytes, g_rayListBuffer);

	thrust::device_ptr<CompressedRay> devicePtr((CompressedRay*)rayListPtr);

	thrust::sort(devicePtr, devicePtr + rayCount);

	hipGraphicsUnmapResources(1, &g_rayListBuffer);
}